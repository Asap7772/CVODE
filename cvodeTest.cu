//
// Created by Anikait Singh on 2019-08-06.
// realtype
//
#include "fblin.h"
#include "jacobian.h"
#include "sundials_nvector.h"
#include "sundials_types.h"
#include "nvector_serial.h"
#include "cvode.h"
#include "sunlinsol_spgmr.h"
#include <hip/hip_runtime.h>
#include <nvector/nvector_cuda.h>
#include <sundials/sundials_math.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdbool.h>
#include <unistd.h>
// typedef hipStream_t hipStream_t;


#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

struct _UserData {
    //constants
    int neq, nr, nphi, nz;
    double rmin, rmax, phimin, phimax, zmin, zmax, eps1, eps2, eps3, delta_phi;
    //static arrays
    double *raxis, *phiaxis, *zaxis, *BR4D, *BZ4D;
    double *raxis_d, *phiaxis_d, *zaxis_d, *BR4D_d, *BZ4D_d;
};

typedef struct _UserData *UserData;
//typedef double realtype;
//typedef double *N_Vector;

/* Problem setup and initialization functions */
static UserData SetUserData(int neq, double rmin, double rmax, double phimin, double phimax, double zmin, double zmax,
                            int nr, int nphi, int nz, double eps1, double eps2, double eps3, double *raxis,
                            double *phiaxis, double *zaxis, double *BR4D, double *BZ4D, double *raxis_d,
                            double *phiaxis_d, double *zaxis_d, double *BR4D_d, double *BZ4D_d, double delta_phi) {

    UserData ud = (UserData) malloc(sizeof(*ud));
//    if (check_retval((void *) ud, "AllocUserData", 2)) return (NULL);
    ud->neq = neq;
    ud->rmin = rmin;
    ud->rmax = rmax;
    ud->phimin = phimin;
    ud->phimax = phimax;
    ud->zmin = zmin;
    ud->zmax = zmax;
    ud->nr = nr;
    ud->nphi = nphi;
    ud->nz = nz;
    ud->eps1 = eps1;
    ud->eps2 = eps2;
    ud->eps3 = eps3;
    ud->raxis = raxis;
    ud->raxis_d = raxis_d;
    ud->zaxis = zaxis;
    ud->zaxis_d = zaxis_d;
    ud->phiaxis = phiaxis;
    ud->phiaxis_d = phiaxis_d;
    ud->BR4D = BR4D;
    ud->BR4D_d = BR4D_d;
    ud->BZ4D = BZ4D;
    ud->BZ4D_d = BZ4D_d;
    ud->delta_phi = delta_phi;
    return ud;
}

/* Functions Called by the Solver */
static int f(realtype t, N_Vector u, N_Vector udot, void *user_data) {
    UserData data = (UserData) user_data;
    //NVECTOR SYNTAX
    //  NON-CUDA
//    double *u_data = NV_DATA_S(u);
//    double *udot_data = NV_DATA_S(udot);
    //  CUDA
    double *u_data_d = N_VGetDeviceArrayPointer_Cuda(u);
    double *u_data = N_VGetHostArrayPointer_Cuda(u);
    double *udot_data_d = N_VGetDeviceArrayPointer_Cuda(udot);
    double *udot_data = N_VGetHostArrayPointer_Cuda(udot);

   rhside_lsode_kernel(t, u_data,u_data_d, udot_data, udot_data_d, data->rmin, data->rmax, data->phimin, data->phimax, data->zmin,
                        data->zmax, data->nr, data->nphi, data->nz, data->eps1, data->eps2, data->eps3, data->raxis,
                        data->phiaxis, data->zaxis, data->BR4D, data->BZ4D, data->raxis_d, data->phiaxis_d, data->zaxis_d,
			data->BR4D_d, data->BZ4D_d, data->delta_phi);
    return 0;
}

static int jtv(N_Vector v, N_Vector Jv, realtype t, N_Vector u, N_Vector fu, void *user_data, N_Vector temp) {
    UserData data = (UserData) user_data;
    //NVECTOR SYNTAX
    //  NON-CUDA
//    double *u_data = NV_DATA_S(u);
//    double *udot_data = NV_DATA_S(fu);
//    double *vec_data = NV_DATA_S(v);
//    double *out_data = NV_DATA_S(Jv);
    //  CUDA
    double *u_data = N_VGetHostArrayPointer_Cuda(u);
    double *u_data_d = N_VGetDeviceArrayPointer_Cuda(u);
    double *udot_data = N_VGetHostArrayPointer_Cuda(fu);
    double *vec_data = N_VGetHostArrayPointer_Cuda(v);
    double *out_data = N_VGetHostArrayPointer_Cuda(Jv);

    int nrpd = data->neq * data->neq;
    double pd[nrpd];

    jacobian_lsode_kernelC(data->neq, t, u_data, u_data_d, pd, nrpd, data->rmin, data->rmax, data->phimin, data->phimax,
                           data->zmin, data->zmax, data->nr, data->nphi, data->nz, data->eps1, data->eps2, data->eps3,
                           data->raxis, data->phiaxis, data->zaxis, data->BR4D, data->BZ4D,data->raxis_d, data->phiaxis_d,
			    data->zaxis_d, data->BR4D_d, data->BZ4D_d, data->delta_phi);

    // 2 x 2 matrix times 2 x 1 vector
    out_data[0] = pd[0] * vec_data[0] + pd[2] * vec_data[1];
    out_data[0] = pd[1] * vec_data[0] + pd[3] * vec_data[1];
    return 0;
}

//method to call from fortran
extern "C"
void evaluatecvode_(int *neq_pointer, double *uval, double *t_pointer, double *tout_pointer, double *reltol_pointer,
                    double *abstol_pointer, double *rmin_pointer, double *rmax_pointer, double *phimin_pointer,
                    double *phimax_pointer, double *zmin_pointer, double *zmax_pointer, int *nr_pointer,
                    int *nphi_pointer, int *nz_pointer, double *eps1_pointer, double *eps2_pointer,
                    double *eps3_pointer, double *raxis, double *phiaxis, double *zaxis, double *BR4D, double *BZ4D,
                    double *delta_phi_pointer) {

    int neq = *neq_pointer;
    double t = *t_pointer;
    double tout = *tout_pointer;
    double reltol = *reltol_pointer;
    double abstol = *abstol_pointer;
    double rmin = *rmin_pointer;
    double rmax = *rmax_pointer;
    double phimin = *phimin_pointer;
    double phimax = *phimax_pointer;
    double zmin = *zmin_pointer;
    double zmax = *zmax_pointer;
    int nr = *nr_pointer;
    int nphi = *nphi_pointer;
    int nz = *nz_pointer;
    double eps1 = *eps1_pointer;
    double eps2 = *eps2_pointer;
    double eps3 = *eps3_pointer;
    double delta_phi = *delta_phi_pointer;

    N_Vector u;
    int iout, retval;
    void *cvode_mem;
    SUNLinearSolver LS;
    UserData data;


    //    ifaccess("a.dat" , F_OK ) == -1 ) {
    //FILE *fp = fopen("a.dat","w+");
    //fprintf(fp, "neq = %d;", neq);
    //fprintf(fp, "neq = %d;", neq);
    //fclose(fp);
    //}
    // fprintf(stderr,"%f %f\n", t, tout);

double *raxis_d, *phiaxis_d, *zaxis_d, *BR4D_d, *BZ4D_d;
 fprintf(stderr, "Before malloc cuda\n");
 fprintf(stderr, "nr = %d, nphi =%d, nz = %d\n", nr, nphi, nz);
//malloc cuda
 fprintf(stderr, "one\n");
 HANDLE_ERROR(hipMalloc(&raxis_d,(size_t)(nr*sizeof(double))));
 fprintf(stderr, "two\n");
 HANDLE_ERROR(hipMalloc(&phiaxis_d,(size_t)(nphi*sizeof(double))));
 fprintf(stderr, "three\n");
 HANDLE_ERROR(hipMalloc(&zaxis_d,(size_t)(nz*sizeof(double))));
 fprintf(stderr, "four\n");
 HANDLE_ERROR(hipMalloc(&BR4D_d,(size_t)(nr*nphi*nz*8*sizeof(double))));
 fprintf(stderr, "five\n");
 HANDLE_ERROR(hipMalloc(&BZ4D_d,(size_t)(nr*nphi*nz*8*sizeof(double))));
 fprintf(stderr, "After malloc cuda\n");
//memcpy cuda
 HANDLE_ERROR(hipMemcpy(raxis_d, raxis, (size_t)(nr * sizeof(double)), hipMemcpyHostToDevice));
 HANDLE_ERROR(hipMemcpy(phiaxis_d, phiaxis, (size_t)(nphi * sizeof(double)), hipMemcpyHostToDevice));
 HANDLE_ERROR(hipMemcpy(zaxis_d, zaxis, (size_t)(nz * sizeof(double)), hipMemcpyHostToDevice));
 HANDLE_ERROR(hipMemcpy(BR4D_d, BR4D, (size_t)(8*nr*nphi*nz * sizeof(double)), hipMemcpyHostToDevice));
 HANDLE_ERROR(hipMemcpy(BZ4D_d, BZ4D, (size_t)(8*nr*nphi*nz * sizeof(double)), hipMemcpyHostToDevice));
 fprintf(stderr, "After memcpy cuda\n");
 data = SetUserData(neq, rmin, rmax, phimin, phimax, zmin, zmax, nr, nphi, nz, eps1, eps2, eps3,raxis, phiaxis, zaxis, BR4D, BZ4D,
 		   raxis_d, phiaxis_d,zaxis_d, BR4D_d, BZ4D_d, delta_phi);
 fprintf(stderr, "After SetUserData\n");
    u = N_VMake_Serial(data->neq, uval);
 double *u_d;
 HANDLE_ERROR(hipMalloc(&u_d, (size_t)(neq * sizeof(double))));
 fprintf(stderr, "before Nvect stuff uval %p u_d %p\n",uval, u_d);
 u = N_VMake_Cuda(neq, uval, u_d);
 fprintf(stderr,"before cvode uval %p u_d %p\n",uval, u_d);
 HANDLE_ERROR(hipMemcpy(u_d, uval, (size_t)(neq * sizeof(double)), hipMemcpyHostToDevice));

 cvode_mem = CVodeCreate(CV_BDF);
 retval = CVodeInit(cvode_mem, f, t, u);
 retval = CVodeSStolerances(cvode_mem, reltol, abstol);
 retval = CVodeSetUserData(cvode_mem, data);
 LS = SUNLinSol_SPGMR(u, PREC_NONE, 0);
 retval = CVodeSetLinearSolver(cvode_mem, LS, NULL);
 retval = CVodeSetJacTimes(cvode_mem, NULL, jtv);

 retval = CVode(cvode_mem, tout, u, &t, CV_NORMAL);
 retval = CVode(cvode_mem, t + delta_phi, u, &tout, CV_NORMAL);
 //double* x = NV_DATA_S(u);
 u_d = N_VGetDeviceArrayPointer_Cuda(u);
 fprintf(stderr,"after cvode  uval %p u_d %p\n",uval, u_d);
 //N_VCopyFromDevice_Cuda(u);
 HANDLE_ERROR(hipMemcpy(uval, u_d, (size_t)(neq * sizeof(double)), hipMemcpyDeviceToHost));
 //    for(int i = 0; i<neq; i++){
 //        uval[i] = x[i];
 //    }
 //    retval = CVodeGetNumSteps(cvode_mem, &nst);
 fprintf(stderr, "leaving cvodeTest\n");
 hipFree(raxis_d);
 hipFree(phiaxis_d);
 hipFree(zaxis_d);
 hipFree(BR4D_d);
 hipFree(BZ4D_d);
 hipFree(u_d);
 free(data);
}
