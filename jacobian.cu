#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "spline.h"
// Used to Calculate Jacobian
// Created by Anikait Singh on 2019-08-01.
//

void jacobian_lsode_kernelC(int neq, double phi, double *q,double *q_d, double *pd, int nrpd, double rmin, double rmax, double phimin,
                            double phimax, double zmin, double zmax, int nr, int nphi, int nz, double eps1, double eps2, double eps3,
                            double *raxis, double *phiaxis,double *zaxis, double *BR4D, double *BZ4D,double *raxis_d,
                            double *phiaxis_d, double *zaxis_d, double *BR4D_d, double *BZ4D_d, double delta_phi) {

    int ier, i, j, k;
    double r_temp, phi_temp, z_temp, xparam,
            yparam, zparam, hx, hy, hz, hxi, hyi, hzi, one = 1;
    double fval[4];
    //int ict[8] = {1, 1, 1, 1, 0, 0, 0, 0};

    ier = 0;
    r_temp = q[0];
    z_temp = q[1];
    phi_temp = fmod(phi, delta_phi);

    if (phi_temp < 0) {
        phi_temp = delta_phi + phi_temp;
    }

    if ((r_temp >= rmin - eps1) && (r_temp <= rmax + eps1) &&
        (phi_temp >= phimin - eps2) && (phi_temp <= phimax + eps2) &&
        (z_temp >= zmin - eps3) && (z_temp <= zmax + eps3)) {
        int count = 0;
        for (int ii = 0; ii < nr; ii++) {
            if (raxis[ii] < r_temp) {
                count++;
            }
        }
        i = fmin(fmax(count, 1), nr - 1);

        count = 0;
        for (int ii = 0; ii < nphi; ii++) {
            if (phiaxis[ii] < phi_temp) {
                count++;
            }
        }
        j = fmin(fmax(count, 1), nphi - 1);

        count = 0;
        for (int ii = 0; ii < nz; ii++) {
            if (zaxis[ii] < z_temp) {
                count++;
            }
        }
        k = fmin(fmax(count, 1), nz - 1);
        hx = raxis[i] - raxis[i - 1];
        hy = phiaxis[j] - phiaxis[j - 1];
        hz = zaxis[k] - zaxis[k - 1];
        hxi = one / hx;
        hyi = one / hy;
        hzi = one / hz;
        xparam = (r_temp - raxis[i - 1]) * hxi;
        yparam = (phi_temp - phiaxis[j - 1]) * hyi;
        zparam = (z_temp - zaxis[k - 1]) * hzi;

        double *fval_d;
        hipMalloc(&fval_d, sizeof(fval));
        hipMemcpy(fval_d, fval, sizeof(fval), hipMemcpyHostToDevice);

        r8herm3fcn<<<1,1>>>(1, 1, fval_d, i, j, k, xparam, yparam, zparam, hx, hxi, hy, hyi, hz, hzi, BR4D_d, nr, nphi, nz);
//        //dBR/dR F had (1,1) transposed for C
        hipMemcpy(fval, fval_d, sizeof(fval), hipMemcpyDeviceToHost);
        pd[0] = fval[1];
//        //dBR/dZ F had (1,2) transposed for C
        pd[2] = fval[3];


        r8herm3fcn<<<1,1>>>(1, 1, fval_d, i, j, k, xparam, yparam, zparam, hx, hxi, hy, hyi, hz, hzi, BZ4D_d, nr, nphi, nz);
        hipMemcpy(fval, fval_d, sizeof(fval), hipMemcpyDeviceToHost);
//        //dBZ/dR F had (2,1) transposed for C
        pd[1] = fval[1];
//        //dBZ/dZ F had (2,2) transposed for C
        pd[3] = fval[3];
        hipFree(fval_d);
    }
}
