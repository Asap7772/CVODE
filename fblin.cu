#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "spline.h"
// Used to Calculate RHS
// Created by Anikait Singh on 2019-08-01.
//

void rhside_lsode_kernel(double phi, double *q, double *q_d, double *qdot, double *qdot_d, double rmin, double rmax,
                         double phimin, double phimax, double zmin, double zmax,
                         int nr, int nphi, int nz, double eps1,
                         double eps2, double eps3, double *raxis, double *phiaxis,
                         double *zaxis, double *BR4D, double *BZ4D,double *raxis_d,
                         double *phiaxis_d, double *zaxis_d, double *BR4D_d, double *BZ4D_d, double delta_phi) {

    const int ivec = 1; //loop of one implemented in case
    int ier;
    double r_temp, phi_temp, z_temp, br_temp, bz_temp, hy, hz, hyi, hzi, one = 1;
    double fval[1], xparam, yparam, zparam, hx, hxi;
    int i, j, k;
    //int ict[8] = {1, 1, 1, 1, 0, 0, 0, 0};

    ier = 0;
    r_temp = q[0];
    z_temp = q[1];
    phi_temp = fmod(phi, delta_phi);

    if (phi_temp < 0) {
        phi_temp = delta_phi + phi_temp;
    }

    br_temp = 0.0;
    bz_temp = 0.0;
    if ((r_temp >= rmin - eps1) && (r_temp <= rmax + eps1) &&
        (phi_temp >= phimin - eps2) && (phi_temp <= phimax + eps2) &&
        (z_temp >= zmin - eps3) && (z_temp <= zmax + eps3)) {
        int count = 0;
        for (int ii = 0; ii < nr; ii++) {
            if (raxis[ii] < r_temp) {
                count++;
            }
        }
        i = fmin(fmax(count, 1), nr - 1);

        count = 0;
        for (int ii = 0; ii < nphi; ii++) {
            if (phiaxis[ii] < phi_temp) {
                count++;
            }
        }
        j = fmin(fmax(count, 1), nphi - 1);

        count = 0;
        for (int ii = 0; ii < nz; ii++) {
            if (zaxis[ii] < z_temp) {
                count++;
            }
        }
        k = fmin(fmax(count, 1), nz - 1);

        hx = raxis[i] - raxis[i - 1];
        hy = phiaxis[j] - phiaxis[j - 1];
        hz = zaxis[k] - zaxis[k - 1];

        hxi = one / hx;
        hyi = one / hy;
        hzi = one / hz;

        xparam = (r_temp - raxis[i - 1]) * hxi;
        yparam = (phi_temp - phiaxis[j - 1]) * hyi;
        zparam = (z_temp - zaxis[k - 1]) * hzi;

        double *fval_d;
        hipMalloc(&fval_d, sizeof(fval));
        hipMemcpy(fval_d, fval, sizeof(fval), hipMemcpyHostToDevice);

        r8herm3fcn<<<1,1>>>(ivec, 1, fval_d, i, j, k, xparam, yparam, zparam, hx, hxi, hy, hyi, hz, hzi, BR4D_d, nr, nphi, nz);
        hipMemcpy(fval, fval_d, sizeof(fval), hipMemcpyDeviceToHost);
        br_temp = fval[0];

        r8herm3fcn<<<1,1>>>(ivec, 1, fval_d, i, j, k, xparam, yparam, zparam, hx, hxi, hy, hyi, hz, hzi, BZ4D_d, nr, nphi, nz);
        hipMemcpy(fval, fval_d, sizeof(fval), hipMemcpyDeviceToHost);
        bz_temp = fval[0];
    }

    qdot[0] = br_temp;
    qdot[1] = bz_temp;
}
