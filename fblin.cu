#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "spline.h"
// Used to Calculate RHS
// Created by Anikait Singh on 2019-08-01.
//

/* double max(double one, double two) { */
/*     return one > two ? one : two; */
/* } */

/* double min(double one, double two) { */
/*     return one > two ? two : one; */
/* } */

/*
void rhside_lsode_kernel_(double *phi_pointer, double *q, double *qdot, double *rmin_pointer, double *rmax_pointer,
                          double *phimin_pointer, double *phimax_pointer, double *zmin_pointer, double *zmax_pointer,
                          int *nr_pointer, int *nphi_pointer, int *nz_pointer, double *eps1_pointer,
                          double *eps2_pointer, double *eps3_pointer, double *raxis, double *phiaxis,
                          double *zaxis, double *BR4D, double *BZ4D, double *delta_phi_pointer) {
    double phi = *phi_pointer;
    double rmin = *rmin_pointer;
    double rmax = *rmax_pointer;
    double phimin = *phimin_pointer;
    double phimax = *phimax_pointer;
    double zmin = *zmin_pointer;
    double zmax = *zmax_pointer;
    int nr = *nr_pointer;
    int nphi = *nphi_pointer;
    int nz = *nz_pointer;
    double eps1 = *eps1_pointer;
    double eps2 = *eps2_pointer;
    double eps3 = *eps3_pointer;
    double delta_phi = *delta_phi_pointer;


    const int ivec = 1; //loop of one implemented in case
    int ier;
    double r_temp, phi_temp, z_temp, br_temp, bz_temp, hy, hz, hyi, hzi, one = 1;
    double fval[1], xparam, yparam, zparam, hx, hxi;
    int i, j, k;
    int ict[8] = {1, 1, 1, 1, 0, 0, 0, 0};

    ier = 0;
    r_temp = q[0];
    z_temp = q[1];
    phi_temp = fmod(phi, delta_phi);

    if (phi_temp < 0) {
        phi_temp = delta_phi + phi_temp;
    }

    br_temp = 0.0;
    bz_temp = 0.0;
    if ((r_temp >= rmin - eps1) && (r_temp <= rmax + eps1) &&
        (phi_temp >= phimin - eps2) && (phi_temp <= phimax + eps2) &&
        (z_temp >= zmin - eps3) && (z_temp <= zmax + eps3)) {
        int count = 0;
        for (int i = 0; i < nr; i++) {
            if (raxis[i] < r_temp) {
                count++;
            }
        }
        i = fmin(fmax(count, 1), nr - 1);

        count = 0;
        for (int i = 0; i < nphi; i++) {
            if (phiaxis[i] < phi_temp) {
                count++;
            }
        }
        j = fmin(fmax(count, 1), nphi - 1);

        count = 0;
        for (int i = 0; i < nz; i++) {
            if (zaxis[i] < z_temp) {
                count++;
            }
        }
        k = fmin(fmax(count, 1), nz - 1);
        hx = raxis[i] - raxis[i - 1];
        hy = phiaxis[j] - phiaxis[j - 1];
        hz = zaxis[k] - zaxis[k - 1];
        hxi = one / hx;
        hyi = one / hy;
        hzi = one / hz;
        xparam = (r_temp - raxis[i - 1]) * hxi;
        yparam = (phi_temp - phiaxis[j - 1]) * hyi;
        zparam = (z_temp - zaxis[k - 1]) * hzi;
        r8herm3fcn(ict, ivec, 1, fval, i, j, k, xparam, yparam, zparam, hx, hxi, hy, hyi, hz, hzi, BR4D, nr, nphi, nz);
        br_temp = fval[0];
        r8herm3fcn(ict, ivec, 1, fval, i, j, k, xparam, yparam, zparam, hx, hxi, hy, hyi, hz, hzi, BZ4D, nr, nphi, nz);
        bz_temp = fval[0];
    }

    qdot[0] = br_temp;
    qdot[1] = bz_temp;
}
*/

void rhside_lsode_kernel(double phi, double *q, double *q_d, double *qdot, double *qdot_d, double rmin, double rmax,
                          double phimin, double phimax, double zmin, double zmax,
                          int nr, int nphi, int nz, double eps1,
                          double eps2, double eps3, double *raxis, double *phiaxis,
                          double *zaxis, double *BR4D, double *BZ4D,double *raxis_d,
			  double *phiaxis_d, double *zaxis_d, double *BR4D_d, double *BZ4D_d, double delta_phi) {

    const int ivec = 1; //loop of one implemented in case
    int ier;
    double r_temp, phi_temp, z_temp, br_temp, bz_temp, hy, hz, hyi, hzi, one = 1;
    double fval[1], xparam, yparam, zparam, hx, hxi;
    int i, j, k;
    //int ict[8] = {1, 1, 1, 1, 0, 0, 0, 0};

    ier = 0;
    r_temp = q[0];
    z_temp = q[1];
    phi_temp = fmod(phi, delta_phi);

    if (phi_temp < 0) {
        phi_temp = delta_phi + phi_temp;
    }

    br_temp = 0.0;
    bz_temp = 0.0;
    if ((r_temp >= rmin - eps1) && (r_temp <= rmax + eps1) &&
        (phi_temp >= phimin - eps2) && (phi_temp <= phimax + eps2) &&
        (z_temp >= zmin - eps3) && (z_temp <= zmax + eps3)) {
        int count = 0;
        for (int ii = 0; ii < nr; ii++) {
            if (raxis[ii] < r_temp) {
                count++;
            }
        }
        i = fmin(fmax(count, 1), nr - 1);

        count = 0;
        for (int ii = 0; ii < nphi; ii++) {
            if (phiaxis[ii] < phi_temp) {
                count++;
            }
        }
        j = fmin(fmax(count, 1), nphi - 1);

        count = 0;
        for (int ii = 0; ii < nz; ii++) {
            if (zaxis[ii] < z_temp) {
                count++;
            }
        }
        k = fmin(fmax(count, 1), nz - 1);

        hx = raxis[i] - raxis[i - 1];
        hy = phiaxis[j] - phiaxis[j - 1];
        hz = zaxis[k] - zaxis[k - 1];

        hxi = one / hx;
        hyi = one / hy;
        hzi = one / hz;

        xparam = (r_temp - raxis[i - 1]) * hxi;
        yparam = (phi_temp - phiaxis[j - 1]) * hyi;
        zparam = (z_temp - zaxis[k - 1]) * hzi;

	double *fval_d;
	hipMalloc(&fval_d, sizeof(fval));
	hipMemcpy(fval_d, fval, sizeof(fval), hipMemcpyHostToDevice);

        r8herm3fcn<<<1,1>>>(ivec, 1, fval_d, i, j, k, xparam, yparam, zparam, hx, hxi, hy, hyi, hz, hzi, BR4D_d, nr, nphi, nz);
	hipMemcpy(fval, fval_d, sizeof(fval), hipMemcpyDeviceToHost);	
        br_temp = fval[0];

        r8herm3fcn<<<1,1>>>(ivec, 1, fval_d, i, j, k, xparam, yparam, zparam, hx, hxi, hy, hyi, hz, hzi, BZ4D_d, nr, nphi, nz);
	hipMemcpy(fval, fval_d, sizeof(fval), hipMemcpyDeviceToHost);		
        bz_temp = fval[0];
    }

    qdot[0] = br_temp;
    qdot[1] = bz_temp;
}
