#include "hip/hip_runtime.h"
#ifdef __NVCC__
__device__
#endif
double transposeLookup4D(double *arr, int i, int j, int k, int l, int s1, int s2, int s3, int s4) {
    //j, k, l are one based
    j = j - 1;
    k = k - 1;
    l = l - 1;
    return arr[l * s3 * s2 * s1 + k * s2 * s1 + j * s1 + i];
}

//ivec = 1
void evaluateSpline(int ivec, int ivecd, double *fval, int i, int j, int k, double xp,
                    double yp, double zp, double hx, double hxi, double hy, double hyi,
                    double hz, double hzi, double *fin, int inf2, int inf3, int nz) {

    double sum = 0;
    int ict[8] = {1, 1, 1, 1, 0, 0, 0, 0};
    int s1 = 8;


    //Currently allocation running serially on device

    //don't know if it would be better to allocate on host and copy or allocate directly on device
    double* constants_d, sum_d;
    const int chunkSize = 8
    dim3 grid_size = {4};
    dim3 block_size = {8};
    const int N = 256;

    hipMalloc(&constants_d, N*sizeof(constants));
    hipMalloc(&sum_d, grid_size.x*block_size.x*sizeof(sum_d));
    allocateArray<<<1,1>>>(xp, yp, zp, hx, hxi, hy, hyi, hz, hzi, constants_d, N);

    summationLookupKernel<<<grid_size,block_size>>(fval, i, j, k, fin, inf2, inf3, nz, constants_d, chunkSize, sum_d);
}

//check if done correctly
#ifdef __NVCC__
__global__
#endif
void summationLookupKernel(double *fval, int i, int j, int k, double *fin, int inf2, int inf3, int nz, double * constants, int chunkSize, double* sum){
    int tid = blockId.x * blockDim.x + threadId.x;
    int i = 0;

    sum[tid] = constants[tid * chunkSize + i++] * transposeLookup4D(fin, threadId.x, i, j, k, s1, inf2, inf3, nz) +
    constants[tid * chunkSize + i++] * transposeLookup4D(fin, threadId.x, i, j + 1, k, s1, inf2, inf3, nz)) +
    constants[tid * chunkSize + i++] * transposeLookup4D(fin, threadId.x, i + 1, j, k, s1, inf2, inf3, nz) +
    constants[tid * chunkSize + i++] * transposeLookup4D(fin, threadId.x, i + 1, j + 1, k, s1, inf2, inf3, nz))) +
    constants[tid * chunkSize + i++] * transposeLookup4D(fin, threadId.x, i, j, k + 1, s1, inf2, inf3, nz) +
    constants[tid * chunkSize + i++] * transposeLookup4D(fin, threadId.x, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
    constants[tid * chunkSize + i++] * transposeLookup4D(fin, threadId.x, i + 1, j, k + 1, s1, inf2, inf3, nz) +
    constants[tid * chunkSize + i++] * transposeLookup4D(fin, threadId.x, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)));

    //
    if(threadId.x == 0){
        fval[blockId.x] = 0;
        for(int i = 0; i< blockDim.x; i++){
            fval[blockId.x] += sum[blockId.x*blockDim.x + i];
        }
    }
}

#ifdef __NVCC__
__global__
#endif
void allocateArray(double xp, double yp, double zp, double hx, double hxi,double hy, double hyi, double hz,
        double hzi, double * constants, const int size){
    int iadr;
    double xpi, xp2, xpi2, ax, axbar, bx, bxbar, ypi, yp2, ypi2, ay;
    double aybar, by, bybar, zpi, zp2, zpi2, az, azbar, bz, bzbar, axp;
    double axbarp, bxp, bxbarp, ayp, aybarp, byp, bybarp, azp, azbarp, bzp;
    double bzbarp;

    //x
    xpi = 1.0 - xp;
    xp2 = xp * xp;
    xpi2 = xpi * xpi;
    ax = xp2 * (3.0 - 2.0 * xp);
    axbar = 1.0 - ax;
    bx = -xp2 * xpi;
    bxbar = xpi2 * xp;

    //y
    ypi = 1.0 - yp;
    yp2 = yp * yp;
    ypi2 = ypi * ypi;
    ay = yp2 * (3.0 - 2.0 * yp);
    aybar = 1.0 - ay;
    by = -yp2 * ypi;
    bybar = ypi2 * yp;

    //z
    zpi = 1.0 - zp;
    zp2 = zp * zp;
    zpi2 = zpi * zpi;
    az = zp2 * (3.0 - 2.0 * zp);
    azbar = 1.0 - az;
    bz = -zp2 * zpi;
    bzbar = zpi2 * zp;

    iadr = 0;

    //derivatives
    axp = 6.0 * xp * xpi;
    axbarp = -axp;
    bxp = xp * (3.0 * xp - 2.0);
    bxbarp = xpi * (3.0 * xpi - 2.0);

    ayp = 6.0 * yp * ypi;
    aybarp = -ayp;
    byp = yp * (3.0 * yp - 2.0);
    bybarp = ypi * (3.0 * ypi - 2.0);

    azp = 6.0 * zp * zpi;
    azbarp = -azp;
    bzp = zp * (3.0 * zp - 2.0);
    bzbarp = zpi * (3.0 * zpi - 2.0);

    int i = 0;

    //---------
    //BLOCK ONE
    //---------

    //thread one
    constants[i++] = azbar * axbar * aybar;
    constants[i++] = azbar * axbar * ay;
    constants[i++] = azbar * ax * aybar;
    constants[i++] = azbar * ax * ay;

    constants[i++] = az * axbar * aybar;
    constants[i++] = az * axbar * ay;
    constants[i++] = az * ax * aybar;
    constants[i++] = az * ax * ay;

    //thread two
    constants[i++] = hx * azbar * bxbar * aybar;
    constants[i++] = hx * azbar * bxbar * ay;
    constants[i++] = hx * azbar * bx * aybar;
    constants[i++] = hx * azbar * bx * ay;

    constants[i++] = hx * az * bxbar * aybar;
    constants[i++] = hx * az * bxbar * ay;
    constants[i++] = hx * az * bx * aybar;
    constants[i++] = hx * az * bx * ay;

    //thread three
    constants[i++] = hy * azbar * axbar * bybar;
    constants[i++] = hy * azbar * axbar * by;
    constants[i++] = hy * azbar * ax * bybar;
    constants[i++] = hy * azbar * ax * by;

    constants[i++] = hy * az * axbar * bybar;
    constants[i++] = hy * az * axbar * by;
    constants[i++] = hy * az * ax * bybar;
    constants[i++] = hy * az * ax * by;

    //thread four
    constants[i++] = hz * bzbar * axbar * aybar;
    constants[i++] = hz * bzbar * axbar * ay;
    constants[i++] = hz * bzbar * ax * aybar;
    constants[i++] = hz * bzbar * ax * ay;

    constants[i++] = hz * bz * axbar * aybar;
    constants[i++] = hz * bz * axbar * ay;
    constants[i++] = hz * bz * ax * aybar;
    constants[i++] = hz * bz * ax * ay;

    //thread five
    constants[i++] = hx * hy * azbar * bxbar * bybar;
    constants[i++] = hx * hy * azbar * bxbar * by;
    constants[i++] = hx * hy * azbar * bx * bybar;
    constants[i++] = hx * hy * azbar * bx * by;

    constants[i++] = hx * hy * az * bxbar * bybar;
    constants[i++] = hx * hy * az * bxbar * by;
    constants[i++] = hx * hy * az * bx * bybar;
    constants[i++] = hx * hy * az * bx * by;

    //thread six
    constants[i++] = hx * hz * bzbar * bxbar * aybar;
    constants[i++] = hx * hz * bzbar * bxbar * ay;
    constants[i++] = hx * hz * bzbar * bx * aybar;
    constants[i++] = hx * hz * bzbar * bx * ay;

    constants[i++] = hx * hz * bz * bxbar * aybar;
    constants[i++] = hx * hz * bz * bxbar * ay;
    constants[i++] = hx * hz * bz * bx * aybar;
    constants[i++] = hx * hz * bz * bx * ay;

    //thread seven
    constants[i++] = hy * hz * bzbar * axbar * bybar;
    constants[i++] = hy * hz * bzbar * axbar * by;
    constants[i++] = hy * hz * bzbar * ax * bybar;
    constants[i++] = hy * hz * bzbar * ax * by;

    constants[i++] = hy * hz * bz * axbar * bybar;
    constants[i++] = hy * hz * bz * axbar * by;
    constants[i++] = hy * hz * bz * ax * bybar;
    constants[i++] = hy * hz * bz * ax * by;

    //thread eight
    constants[i++] = hx * hy * hz * bzbar * bxbar * bybar;
    constants[i++] = hx * hy * hz * bzbar * bxbar * by;
    constants[i++] = hx * hy * hz * bzbar * bx * bybar;
    constants[i++] = hx * hy * hz * bzbar * bx * by;

    constants[i++] = hx * hy * hz * bz * bxbar * bybar;
    constants[i++] = hx * hy * hz * bz * bxbar * by;
    constants[i++] = hx * hy * hz * bz * bx * bybar;
    constants[i++] = hx * hy * hz * bz * bx * by;

    //---------
    //BLOCK TWO
    //---------

    //thread one
    constants[i++] = hxi * azbar * axbarp * aybar;
    constants[i++] = hxi * azbar * axbarp * ay;
    constants[i++] = hxi * azbar * axp * aybar;
    constants[i++] = hxi * azbar * axp * ay;

    constants[i++] = hxi * az * axbarp * aybar;
    constants[i++] = hxi * az * axbarp * ay;
    constants[i++] = hxi * az * axp * aybar;
    constants[i++] = hxi * az * axp * ay;

    //thread two
    constants[i++] = azbar * bxbarp * aybar;
    constants[i++] = azbar * bxbarp * ay;
    constants[i++] = azbar * bxp * aybar;
    constants[i++] = azbar * bxp * ay;

    constants[i++] = az * bxbarp * aybar;
    constants[i++] = az * bxbarp * ay;
    constants[i++] = az * bxp * aybar;
    constants[i++] = az * bxp * ay;

    //thread three
    constants[i++] = hxi * hy * azbar * axbarp * bybar;
    constants[i++] = hxi * hy * azbar * axbarp * by;
    constants[i++] = hxi * hy * azbar * axp * bybar;
    constants[i++] = hxi * hy * azbar * axp * by;

    constants[i++] = hxi * hy * az * axbarp * bybar;
    constants[i++] = hxi * hy * az * axbarp * by;
    constants[i++] = hxi * hy * az * axp * bybar;
    constants[i++] = hxi * hy * az * axp * by;

    //thread four
    constants[i++] = hxi * hz * bzbar * axbarp * aybar;
    constants[i++] = hxi * hz * bzbar * axbarp * ay;
    constants[i++] = hxi * hz * bzbar * axp * aybar;
    constants[i++] = hxi * hz * bzbar * axp * ay;

    constants[i++] = hxi * hz * bz * axbarp * ay;
    constants[i++] = hxi * hz * bz * axbarp * aybar;
    constants[i++] = hxi * hz * bz * axp * aybar;
    constants[i++] = hxi * hz * bz * axp * ay;

    //thread five
    constants[i++] = hy * azbar * bxbarp * bybar;
    constants[i++] = hy * azbar * bxbarp * by;
    constants[i++] = hy * azbar * bxp * bybar;
    constants[i++] = hy * azbar * bxp * by;

    constants[i++] = hy * az * bxbarp * bybar;
    constants[i++] = hy * az * bxbarp * by;
    constants[i++] = hy * az * bxp * bybar;
    constants[i++] = hy * az * bxp * by;

    //thread six
    constants[i++] = hz * bzbar * bxbarp * aybar;
    constants[i++] = hz * bzbar * bxbarp * ay;
    constants[i++] = hz * bzbar * bxp * aybar;
    constants[i++] = hz * bzbar * bxp * ay;

    constants[i++] = hz * bz * bxbarp * aybar;
    constants[i++] = hz * bz * bxbarp * ay;
    constants[i++] = hz * bz * bxp * aybar;
    constants[i++] = hz * bz * bxp * ay;

    //thread seven
    constants[i++] = hxi * hy * hz * bzbar * axbarp * bybar;
    constants[i++] = hxi * hy * hz * bzbar * axbarp * by;
    constants[i++] = hxi * hy * hz * bzbar * axp * bybar;
    constants[i++] = hxi * hy * hz * bzbar * axp * by;

    constants[i++] = hxi * hy * hz * bz * axbarp * bybar;
    constants[i++] = hxi * hy * hz * bz * axbarp * by;
    constants[i++] = hxi * hy * hz * bz * axp * bybar;
    constants[i++] = hxi * hy * hz * bz * axp * by;

    //thread eight
    constants[i++] = hy * hz * bzbar * bxbarp * bybar;
    constants[i++] = hy * hz * bzbar * bxbarp * by;
    constants[i++] = hy * hz * bzbar * bx * bybar;
    constants[i++] = hy * hz * bzbar * bxp * by;

    constants[i++] = hy * hz * bz * bxbarp * bybar;
    constants[i++] = hy * hz * bz * bxbarp * by;
    constants[i++] = hy * hz * bz * bxp * bybar;
    constants[i++] = hy * hz * bz * bxp * by;

    //---------
    //BLOCK THREE
    //---------

    //thread one
    constants[i++] = hyi * azbar * axbar * aybarp;
    constants[i++] = hyi * azbar * axbar * ayp;
    constants[i++] = hyi * azbar * axp * aybarp;
    constants[i++] = hyi * azbar * axp * ayp;

    constants[i++] = hyi * az * axbar * aybarp;
    constants[i++] = hyi * az * axbar * ayp;
    constants[i++] = hyi * az * axp * aybarp;
    constants[i++] = hyi * az * axp * ayp;

    //thread two
    constants[i++] = hyi * hx * azbar * bxbar * aybarp;
    constants[i++] = hyi * hx * azbar * bxbar * ayp;
    constants[i++] = hyi * hx * azbar * bxp * aybarp;
    constants[i++] = hyi * hx * azbar * bxp * ayp;

    constants[i++] = hyi * hx * az * bxbar * aybarp;
    constants[i++] = hyi * hx * az * bxbar * ayp;
    constants[i++] = hyi * hx * az * bxp * aybarp;
    constants[i++] = hyi * hx * az * bxp * ayp;

    //thread three
    constants[i++] = azbar * axbar * bybarp;
    constants[i++] = azbar * axbar * byp;
    constants[i++] = azbar * axp * bybarp;
    constants[i++] = azbar * axp * byp;

    constants[i++] = az * axbar * bybarp;
    constants[i++] = az * axbar * byp;
    constants[i++] = az * axp * bybarp;
    constants[i++] = az * axp * byp;

    //thread four
    constants[i++] = hyi * hz * bzbar * axbar * aybarp;
    constants[i++] = hyi * hz * bzbar * axbar * ayp;
    constants[i++] = hyi * hz * bzbar * axp * aybarp;
    constants[i++] = hyi * hz * bzbar * axp * ayp;

    constants[i++] = hyi * hz * bz * axbar * ayp;
    constants[i++] = hyi * hz * bz * axbar * aybarp;
    constants[i++] = hyi * hz * bz * axp * aybarp;
    constants[i++] = hyi * hz * bz * axp * ayp;

    //thread five
    constants[i++] = hx * azbar * bxbar * bybar;
    constants[i++] = hx * azbar * bxbar * by;
    constants[i++] = hx * azbar * bxp * bybar;
    constants[i++] = hx * azbar * bxp * by;

    constants[i++] = hx * az * bxbar * bybar;
    constants[i++] = hx * az * bxbar * by;
    constants[i++] = hx * az * bxp * bybar;
    constants[i++] = hx * az * bxp * by;

    //thread six
    constants[i++] = hx * hyi * hz * bzbar * bxbar * aybar;
    constants[i++] = hx * hyi * hz * bzbar * bxbar * ay;
    constants[i++] = hx * hyi * hz * bzbar * bxp * aybar;
    constants[i++] = hx * hyi * hz * bzbar * bxp * ay;

    constants[i++] = hx * hyi * hz * bz * bxbar * aybar;
    constants[i++] = hx * hyi * hz * bz * bxbar * ay;
    constants[i++] = hx * hyi * hz * bz * bxp * aybar;
    constants[i++] = hx * hyi * hz * bz * bxp * ay;

    //thread seven
    constants[i++] = hz * bzbar * axbar * bybar;
    constants[i++] = hz * bzbar * axbar * by;
    constants[i++] = hz * bzbar * axp * bybar;
    constants[i++] = hz * bzbar * axp * by;

    constants[i++] = hz * bz * axbar * bybar;
    constants[i++] = hz * bz * axbar * by;
    constants[i++] = hz * bz * axp * bybar;
    constants[i++] = hz * bz * axp * by;

    //thread eight
    constants[i++] = hx * hz * bzbar * bxbar * bybar;
    constants[i++] = hx * hz * bzbar * bxbar * by;
    constants[i++] = hx * hz * bzbar * bx * bybar;
    constants[i++] = hx * hz * bzbar * bxp * by;

    constants[i++] = hx * hz * bz * bxbar * bybar;
    constants[i++] = hx * hz * bz * bxbar * by;
    constants[i++] = hx * hz * bz * bxp * bybar;
    constants[i++] = hx * hz * bz * bxp * by;

    //---------
    //BLOCK FOUR
    //---------

    //thread one
    constants[i++] = hzi * azbarp * axbar * aybar;
    constants[i++] = hzi * azbarp * axbar * ay;
    constants[i++] = hzi * azbarp * ax * aybar;
    constants[i++] = hzi * azbarp * ax * ay;

    constants[i++] = hzi * azp * axbar * aybar;
    constants[i++] = hzi * azp * axbar * ay;
    constants[i++] = hzi * azp * ax * aybar;
    constants[i++] = hzi * azp * ax * ay;

    //thread two
    constants[i++] = hzi * hx * azbarp * bxbar * aybar;
    constants[i++] = hzi * hx * azbarp * bxbar * ay;
    constants[i++] = hzi * hx * azbarp * bx * aybar;
    constants[i++] = hzi * hx * azbarp * bx * ay;

    constants[i++] = hzi * hx * azp * bxbar * aybar;
    constants[i++] = hzi * hx * azp * bxbar * ay;
    constants[i++] = hzi * hx * azp * bx * aybar;
    constants[i++] = hzi * hx * azp * bx * ay;

    //thread three
    constants[i++] = hzi * hy * azbarp * axbar * bybar;
    constants[i++] = hzi * hy * azbarp * axbar * by;
    constants[i++] = hzi * hy * azbarp * ax * bybar;
    constants[i++] = hzi * hy * azbarp * ax * by;

    constants[i++] = hzi * hy * azp * axbar * bybar;
    constants[i++] = hzi * hy * azp * axbar * by;
    constants[i++] = hzi * hy * azp * ax * bybar;
    constants[i++] = hzi * hy * azp * ax * by;

    //thread four
    constants[i++] = bzbarp * axbar * aybar;
    constants[i++] = bzbarp * axbar * ay;
    constants[i++] = bzbarp * ax * aybar;
    constants[i++] = bzbarp * ax * ay;

    constants[i++] = bzp * axbar * aybar;
    constants[i++] = bzp * axbar * ay;
    constants[i++] = bzp * ax * aybar;
    constants[i++] = bzp * ax * ay;

    //thread five
    constants[i++] = hzi * hx * hy * azbarp * bxbar * bybar;
    constants[i++] = hzi * hx * hy * azbarp * bxbar * by;
    constants[i++] = hzi * hx * hy * azbarp * bx * bybar;
    constants[i++] = hzi * hx * hy * azbarp * bx * by;

    constants[i++] = hzi * hx * hy * azp * bxbar * bybar;
    constants[i++] = hzi * hx * hy * azp * bxbar * by;
    constants[i++] = hzi * hx * hy * azp * bx * bybar;
    constants[i++] = hzi * hx * hy * azp * bx * by;

    //thread six
    constants[i++] = hx * bzbarp * bxbar * aybar;
    constants[i++] = hx * bzbarp * bxbar * ay;
    constants[i++] = hx * bzbarp * bx * aybar;
    constants[i++] = hx * bzbarp * bx * ay;

    constants[i++] = hx * bzp * bxbar * aybar;
    constants[i++] = hx * bzp * bxbar * ay;
    constants[i++] = hx * bzp * bx * aybar;
    constants[i++] = hx * bzp * bx * ay;

    //thread seven
    constants[i++] = hy * bzbarp * axbar * bybar;
    constants[i++] = hy * bzbarp * axbar * by;
    constants[i++] = hy * bzbarp * ax * bybar;
    constants[i++] = hy * bzbarp * ax * by;

    constants[i++] = hy * bzp * axbar * bybar;
    constants[i++] = hy * bzp * axbar * by;
    constants[i++] = hy * bzp * ax * bybar;
    constants[i++] = hy * bzp * ax * by;

    //thread eight
    constants[i++] = hx * hy * bzbarp * bxbar * bybar;
    constants[i++] = hx * hy * bzbarp * bxbar * by;
    constants[i++] = hx * hy * bzbarp * bx * bybar;
    constants[i++] = hx * hy * bzbarp * bx * by;

    constants[i++] = hx * hy * bzp * bxbar * bybar;
    constants[i++] = hx * hy * bzp * bxbar * by;
    constants[i++] = hx * hy * bzp * bx * bybar;
    constants[i++] = hx * hy * bzp * bx * by;
}
