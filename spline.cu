
#include <hip/hip_runtime.h>
//
// Created by Anikait Singh on 2019-08-01.
// for interpolation
//


//arr[i][j] to arr[j][i]
//r is rows
//c is columns
//static double transposeLookup2D(double *arr, int i, int j, int r, int c) {
//    return arr[j * r + i];
//}

//arr[i][j][k][l] to arr[l][k][j][i]
//s1,s2,s3,s4 is size of i, j, k, l respetively
#ifdef __NVCC__
__device__
#endif
double transposeLookup4D(double *arr, int i, int j, int k, int l, int s1, int s2, int s3, int s4) {
    //j, k, l are one based
    j = j - 1;
    k = k - 1;
    l = l - 1;
    return arr[l * s3 * s2 * s1 + k * s2 * s1 + j * s1 + i];
}

//ivec = 1
#ifdef __NVCC__
__global__
#endif
void r8herm3fcn(int ivec, int ivecd, double *fval, int i, int j, int k, double xp,
                double yp, double zp, double hx, double hxi, double hy, double hyi,
                double hz, double hzi, double *fin, int inf2, int inf3,
                int nz) {

    int iadr;
    double xpi, xp2, xpi2, ax, axbar, bx, bxbar, ypi, yp2, ypi2, ay;
    double aybar, by, bybar, zpi, zp2, zpi2, az, azbar, bz, bzbar, axp;
    double axbarp, bxp, bxbarp, ayp, aybarp, byp, bybarp, azp, azbarp, bzp;
    double bzbarp;

    double sum = 0;
    int ict[8] = {1,1,1,1,0,0,0,0};

    //x
    xpi = 1.0 - xp;
    xp2 = xp * xp;
    xpi2 = xpi * xpi;
    ax = xp2 * (3.0 - 2.0 * xp);
    axbar = 1.0 - ax;
    bx = -xp2 * xpi;
    bxbar = xpi2 * xp;

    //y
    ypi = 1.0 - yp;
    yp2 = yp * yp;
    ypi2 = ypi * ypi;
    ay = yp2 * (3.0 - 2.0 * yp);
    aybar = 1.0 - ay;
    by = -yp2 * ypi;
    bybar = ypi2 * yp;

    //z
    zpi = 1.0 - zp;
    zp2 = zp * zp;
    zpi2 = zpi * zpi;
    az = zp2 * (3.0 - 2.0 * zp);
    azbar = 1.0 - az;
    bz = -zp2 * zpi;
    bzbar = zpi2 * zp;

    iadr = 0;

    //derivatives
    axp = 6.0 * xp * xpi;
    axbarp = -axp;
    bxp = xp * (3.0 * xp - 2.0);
    bxbarp = xpi * (3.0 * xpi - 2.0);

    ayp = 6.0 * yp * ypi;
    aybarp = -ayp;
    byp = yp * (3.0 * yp - 2.0);
    bybarp = ypi * (3.0 * ypi - 2.0);

    azp = 6.0 * zp * zpi;
    azbarp = -azp;
    bzp = zp * (3.0 * zp - 2.0);
    bzbarp = zpi * (3.0 * zpi - 2.0);

    int s1 = 8;
    if (ict[0] == 1) {
        // iadr = iadr + 1;
        sum = azbar * (
                axbar * (aybar * transposeLookup4D(fin, 0, i, j, k, s1, inf2, inf3, nz) +
                         ay * transposeLookup4D(fin, 0, i, j + 1, k, s1, inf2, inf3, nz)) +
                ax * (aybar * transposeLookup4D(fin, 0, i + 1, j, k, s1, inf2, inf3, nz) +
                      ay * transposeLookup4D(fin, 0, i + 1, j + 1, k, s1, inf2, inf3, nz))) +
              +az * (
                      axbar * (aybar * transposeLookup4D(fin, 0, i, j, k + 1, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 0, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                      ax * (aybar * transposeLookup4D(fin, 0, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                            ay * transposeLookup4D(fin, 0, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)));

        sum = sum + hx * (
                azbar * (
                        bxbar * (aybar * transposeLookup4D(fin, 1, i, j, k, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 1, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (aybar * transposeLookup4D(fin, 1, i + 1, j, k, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 1, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        bxbar * (aybar * transposeLookup4D(fin, 1, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 1, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (aybar * transposeLookup4D(fin, 1, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 1, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hy * (
                azbar * (
                        axbar * (bybar * transposeLookup4D(fin, 2, i, j, k, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 2, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (bybar * transposeLookup4D(fin, 2, i + 1, j, k, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 2, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        axbar * (bybar * transposeLookup4D(fin, 2, i, j, k + 1, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 2, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (bybar * transposeLookup4D(fin, 2, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 2, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hz * (
                bzbar * (
                        axbar * (aybar * transposeLookup4D(fin, 3, i, j, k, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 3, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (aybar * transposeLookup4D(fin, 3, i + 1, j, k, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 3, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        axbar * (aybar * transposeLookup4D(fin, 3, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 3, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (aybar * transposeLookup4D(fin, 3, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 3, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hx * hy * (
                azbar * (
                        bxbar * (bybar * transposeLookup4D(fin, 4, i, j, k, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 4, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (bybar * transposeLookup4D(fin, 4, i + 1, j, k, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 4, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        bxbar * (bybar * transposeLookup4D(fin, 4, i, j, k + 1, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 4, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (bybar * transposeLookup4D(fin, 4, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 4, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hx * hz * (
                bzbar * (
                        bxbar * (aybar * transposeLookup4D(fin, 5, i, j, k, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 5, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (aybar * transposeLookup4D(fin, 5, i + 1, j, k, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 5, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        bxbar * (aybar * transposeLookup4D(fin, 5, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 5, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (aybar * transposeLookup4D(fin, 5, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 5, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hy * hz * (
                bzbar * (
                        axbar * (bybar * transposeLookup4D(fin, 6, i, j, k, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 6, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (bybar * transposeLookup4D(fin, 6, i + 1, j, k, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 6, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        axbar * (bybar * transposeLookup4D(fin, 6, i, j, k + 1, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 6, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (bybar * transposeLookup4D(fin, 6, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 6, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hx * hy * hz * (
                bzbar * (
                        bxbar * (bybar * transposeLookup4D(fin, 7, i, j, k, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 7, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (bybar * transposeLookup4D(fin, 7, i + 1, j, k, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 7, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        bxbar * (bybar * transposeLookup4D(fin, 7, i, j, k + 1, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 7, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (bybar * transposeLookup4D(fin, 7, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 7, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        //v is 0 so ignore
        fval[iadr] = sum;
    }

    if (ict[1] == 1) {
        iadr = iadr + 1;

        sum = hxi * (
                azbar * (
                        axbarp * (aybar * transposeLookup4D(fin, 0, i, j, k, s1, inf2, inf3, nz) +
                                  ay * transposeLookup4D(fin, 0, i, j + 1, k, s1, inf2, inf3, nz)) +
                        axp * (aybar * transposeLookup4D(fin, 0, i + 1, j, k, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 0, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        axbarp * (aybar * transposeLookup4D(fin, 0, i, j, k + 1, s1, inf2, inf3, nz) +
                                  ay * transposeLookup4D(fin, 0, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        axp * (aybar * transposeLookup4D(fin, 0, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 0, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + (
                azbar * (
                        bxbarp * (aybar * transposeLookup4D(fin, 1, i, j, k, s1, inf2, inf3, nz) +
                                  ay * transposeLookup4D(fin, 1, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bxp * (aybar * transposeLookup4D(fin, 1, i + 1, j, k, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 1, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        bxbarp * (aybar * transposeLookup4D(fin, 1, i, j, k + 1, s1, inf2, inf3, nz) +
                                  ay * transposeLookup4D(fin, 1, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bxp * (aybar * transposeLookup4D(fin, 1, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 1, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hxi * hy * (
                azbar * (
                        axbarp * (bybar * transposeLookup4D(fin, 2, i, j, k, s1, inf2, inf3, nz) +
                                  by * transposeLookup4D(fin, 2, i, j + 1, k, s1, inf2, inf3, nz)) +
                        axp * (bybar * transposeLookup4D(fin, 2, i + 1, j, k, s1, inf2, inf3, nz) +
                               by * transposeLookup4D(fin, 2, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        axbarp * (bybar * transposeLookup4D(fin, 2, i, j, k + 1, s1, inf2, inf3, nz) +
                                  by * transposeLookup4D(fin, 2, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        axp * (bybar * transposeLookup4D(fin, 2, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                               by * transposeLookup4D(fin, 2, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hxi * hz * (
                bzbar * (
                        axbarp * (aybar * transposeLookup4D(fin, 3, i, j, k, s1, inf2, inf3, nz) +
                                  ay * transposeLookup4D(fin, 3, i, j + 1, k, s1, inf2, inf3, nz)) +
                        axp * (aybar * transposeLookup4D(fin, 3, i + 1, j, k, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 3, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        axbarp * (aybar * transposeLookup4D(fin, 3, i, j, k + 1, s1, inf2, inf3, nz) +
                                  ay * transposeLookup4D(fin, 3, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        axp * (aybar * transposeLookup4D(fin, 3, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 3, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hy * (
                azbar * (
                        bxbarp * (bybar * transposeLookup4D(fin, 4, i, j, k, s1, inf2, inf3, nz) +
                                  by * transposeLookup4D(fin, 4, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bxp * (bybar * transposeLookup4D(fin, 4, i + 1, j, k, s1, inf2, inf3, nz) +
                               by * transposeLookup4D(fin, 4, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        bxbarp * (bybar * transposeLookup4D(fin, 4, i, j, k + 1, s1, inf2, inf3, nz) +
                                  by * transposeLookup4D(fin, 4, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bxp * (bybar * transposeLookup4D(fin, 4, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                               by * transposeLookup4D(fin, 4, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hz * (
                bzbar * (
                        bxbarp * (aybar * transposeLookup4D(fin, 5, i, j, k, s1, inf2, inf3, nz) +
                                  ay * transposeLookup4D(fin, 5, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bxp * (aybar * transposeLookup4D(fin, 5, i + 1, j, k, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 5, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        bxbarp * (aybar * transposeLookup4D(fin, 5, i, j, k + 1, s1, inf2, inf3, nz) +
                                  ay * transposeLookup4D(fin, 5, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bxp * (aybar * transposeLookup4D(fin, 5, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                               ay * transposeLookup4D(fin, 5, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hxi * hy * hz * (
                bzbar * (
                        axbarp * (bybar * transposeLookup4D(fin, 6, i, j, k, s1, inf2, inf3, nz) +
                                  by * transposeLookup4D(fin, 6, i, j + 1, k, s1, inf2, inf3, nz)) +
                        axp * (bybar * transposeLookup4D(fin, 6, i + 1, j, k, s1, inf2, inf3, nz) +
                               by * transposeLookup4D(fin, 6, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        axbarp * (bybar * transposeLookup4D(fin, 6, i, j, k + 1, s1, inf2, inf3, nz) +
                                  by * transposeLookup4D(fin, 6, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        axp * (bybar * transposeLookup4D(fin, 6, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                               by * transposeLookup4D(fin, 6, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        sum = sum + hy * hz * (
                bzbar * (
                        bxbarp * (bybar * transposeLookup4D(fin, 7, i, j, k, s1, inf2, inf3, nz) +
                                  by * transposeLookup4D(fin, 7, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bxp * (bybar * transposeLookup4D(fin, 7, i + 1, j, k, s1, inf2, inf3, nz) +
                               by * transposeLookup4D(fin, 7, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        bxbarp * (bybar * transposeLookup4D(fin, 7, i, j, k + 1, s1, inf2, inf3, nz) +
                                  by * transposeLookup4D(fin, 7, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bxp * (bybar * transposeLookup4D(fin, 7, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                               by * transposeLookup4D(fin, 7, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );

        fval[iadr] = sum;
    }

    if (ict[2] == 1) {
        iadr = iadr + 1;
        sum = hyi * (
                azbar * (
                        axbar * (aybarp * transposeLookup4D(fin, 0, i, j, k, s1, inf2, inf3, nz) +
                                 ayp * transposeLookup4D(fin, 0, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (aybarp * transposeLookup4D(fin, 0, i + 1, j, k, s1, inf2, inf3, nz) +
                              ayp * transposeLookup4D(fin, 0, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        axbar * (aybarp * transposeLookup4D(fin, 0, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ayp * transposeLookup4D(fin, 0, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (aybarp * transposeLookup4D(fin, 0, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ayp * transposeLookup4D(fin, 0, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hyi * hx * (
                azbar * (
                        bxbar * (aybarp * transposeLookup4D(fin, 1, i, j, k, s1, inf2, inf3, nz) +
                                 ayp * transposeLookup4D(fin, 1, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (aybarp * transposeLookup4D(fin, 1, i + 1, j, k, s1, inf2, inf3, nz) +
                              ayp * transposeLookup4D(fin, 1, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        bxbar * (aybarp * transposeLookup4D(fin, 1, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ayp * transposeLookup4D(fin, 1, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (aybarp * transposeLookup4D(fin, 1, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ayp * transposeLookup4D(fin, 1, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + (
                azbar * (
                        axbar * (bybarp * transposeLookup4D(fin, 2, i, j, k, s1, inf2, inf3, nz) +
                                 byp * transposeLookup4D(fin, 2, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (bybarp * transposeLookup4D(fin, 2, i + 1, j, k, s1, inf2, inf3, nz) +
                              byp * transposeLookup4D(fin, 2, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        axbar * (bybarp * transposeLookup4D(fin, 2, i, j, k + 1, s1, inf2, inf3, nz) +
                                 byp * transposeLookup4D(fin, 2, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (bybarp * transposeLookup4D(fin, 2, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              byp * transposeLookup4D(fin, 2, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hyi * hz * (
                bzbar * (
                        axbar * (aybarp * transposeLookup4D(fin, 3, i, j, k, s1, inf2, inf3, nz) +
                                 ayp * transposeLookup4D(fin, 3, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (aybarp * transposeLookup4D(fin, 3, i + 1, j, k, s1, inf2, inf3, nz) +
                              ayp * transposeLookup4D(fin, 3, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        axbar * (aybarp * transposeLookup4D(fin, 3, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ayp * transposeLookup4D(fin, 3, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (aybarp * transposeLookup4D(fin, 3, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ayp * transposeLookup4D(fin, 3, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hx * (
                azbar * (
                        bxbar * (bybarp * transposeLookup4D(fin, 4, i, j, k, s1, inf2, inf3, nz) +
                                 byp * transposeLookup4D(fin, 4, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (bybarp * transposeLookup4D(fin, 4, i + 1, j, k, s1, inf2, inf3, nz) +
                              byp * transposeLookup4D(fin, 4, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + az * (
                        bxbar * (bybarp * transposeLookup4D(fin, 4, i, j, k + 1, s1, inf2, inf3, nz) +
                                 byp * transposeLookup4D(fin, 4, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (bybarp * transposeLookup4D(fin, 4, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              byp * transposeLookup4D(fin, 4, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hx * hyi * hz * (
                bzbar * (
                        bxbar * (aybarp * transposeLookup4D(fin, 5, i, j, k, s1, inf2, inf3, nz) +
                                 ayp * transposeLookup4D(fin, 5, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (aybarp * transposeLookup4D(fin, 5, i + 1, j, k, s1, inf2, inf3, nz) +
                              ayp * transposeLookup4D(fin, 5, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        bxbar * (aybarp * transposeLookup4D(fin, 5, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ayp * transposeLookup4D(fin, 5, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (aybarp * transposeLookup4D(fin, 5, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ayp * transposeLookup4D(fin, 5, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hz * (
                bzbar * (
                        axbar * (bybarp * transposeLookup4D(fin, 6, i, j, k, s1, inf2, inf3, nz) +
                                 byp * transposeLookup4D(fin, 6, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (bybarp * transposeLookup4D(fin, 6, i + 1, j, k, s1, inf2, inf3, nz) +
                              byp * transposeLookup4D(fin, 6, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        axbar * (bybarp * transposeLookup4D(fin, 6, i, j, k + 1, s1, inf2, inf3, nz) +
                                 byp * transposeLookup4D(fin, 6, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (bybarp * transposeLookup4D(fin, 6, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              byp * transposeLookup4D(fin, 6, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hx * hz * (
                bzbar * (
                        bxbar * (bybarp * transposeLookup4D(fin, 7, i, j, k, s1, inf2, inf3, nz) +
                                 byp * transposeLookup4D(fin, 7, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (bybarp * transposeLookup4D(fin, 7, i + 1, j, k, s1, inf2, inf3, nz) +
                              byp * transposeLookup4D(fin, 7, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bz * (
                        bxbar * (bybarp * transposeLookup4D(fin, 7, i, j, k + 1, s1, inf2, inf3, nz) +
                                 byp * transposeLookup4D(fin, 7, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (bybarp * transposeLookup4D(fin, 7, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              byp * transposeLookup4D(fin, 7, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        fval[iadr] = sum;
    }
    if (ict[3] == 1) {
        iadr = iadr + 1;
        sum = hzi * (
                azbarp * (
                        axbar * (aybar * transposeLookup4D(fin, 0, i, j, k, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 0, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (aybar * transposeLookup4D(fin, 0, i + 1, j, k, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 0, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + azp * (
                        axbar * (aybar * transposeLookup4D(fin, 0, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 0, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (aybar * transposeLookup4D(fin, 0, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 0, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hzi * hx * (
                azbarp * (
                        bxbar * (aybar * transposeLookup4D(fin, 1, i, j, k, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 1, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (aybar * transposeLookup4D(fin, 1, i + 1, j, k, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 1, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + azp * (
                        bxbar * (aybar * transposeLookup4D(fin, 1, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 1, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (aybar * transposeLookup4D(fin, 1, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 1, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hzi * hy * (
                azbarp * (
                        axbar * (bybar * transposeLookup4D(fin, 2, i, j, k, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 2, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (bybar * transposeLookup4D(fin, 2, i + 1, j, k, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 2, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + azp * (
                        axbar * (bybar * transposeLookup4D(fin, 2, i, j, k + 1, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 2, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (bybar * transposeLookup4D(fin, 2, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 2, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + (
                bzbarp * (
                        axbar * (aybar * transposeLookup4D(fin, 3, i, j, k, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 3, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (aybar * transposeLookup4D(fin, 3, i + 1, j, k, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 3, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bzp * (
                        axbar * (aybar * transposeLookup4D(fin, 3, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 3, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (aybar * transposeLookup4D(fin, 3, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 3, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hzi * hx * hy * (
                azbarp * (
                        bxbar * (bybar * transposeLookup4D(fin, 4, i, j, k, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 4, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (bybar * transposeLookup4D(fin, 4, i + 1, j, k, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 4, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + azp * (
                        bxbar * (bybar * transposeLookup4D(fin, 4, i, j, k + 1, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 4, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (bybar * transposeLookup4D(fin, 4, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 4, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hx * (
                bzbarp * (
                        bxbar * (aybar * transposeLookup4D(fin, 5, i, j, k, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 5, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (aybar * transposeLookup4D(fin, 5, i + 1, j, k, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 5, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bzp * (
                        bxbar * (aybar * transposeLookup4D(fin, 5, i, j, k + 1, s1, inf2, inf3, nz) +
                                 ay * transposeLookup4D(fin, 5, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (aybar * transposeLookup4D(fin, 5, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              ay * transposeLookup4D(fin, 5, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hy * (
                bzbarp * (
                        axbar * (bybar * transposeLookup4D(fin, 6, i, j, k, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 6, i, j + 1, k, s1, inf2, inf3, nz)) +
                        ax * (bybar * transposeLookup4D(fin, 6, i + 1, j, k, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 6, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bzp * (
                        axbar * (bybar * transposeLookup4D(fin, 6, i, j, k + 1, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 6, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        ax * (bybar * transposeLookup4D(fin, 6, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 6, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        sum = sum + hx * hy * (
                bzbarp * (
                        bxbar * (bybar * transposeLookup4D(fin, 7, i, j, k, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 7, i, j + 1, k, s1, inf2, inf3, nz)) +
                        bx * (bybar * transposeLookup4D(fin, 7, i + 1, j, k, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 7, i + 1, j + 1, k, s1, inf2, inf3, nz)))
                + bzp * (
                        bxbar * (bybar * transposeLookup4D(fin, 7, i, j, k + 1, s1, inf2, inf3, nz) +
                                 by * transposeLookup4D(fin, 7, i, j + 1, k + 1, s1, inf2, inf3, nz)) +
                        bx * (bybar * transposeLookup4D(fin, 7, i + 1, j, k + 1, s1, inf2, inf3, nz) +
                              by * transposeLookup4D(fin, 7, i + 1, j + 1, k + 1, s1, inf2, inf3, nz)))
        );
        fval[iadr] = sum;
    }
}
